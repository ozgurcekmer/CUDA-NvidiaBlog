#include "hip/hip_runtime.h"
#include "../include/GpuSequential.h"

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuSequential(T* __restrict__ x, T* __restrict__ y, const T MEAN, const T SIGMA, const size_t N)
{
    
}

template<typename T>
void GpuSequential<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dX, BYTES);
    gpuMalloc(&dY, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuSequential<T>::copyH2D()
{
    //gpuMemcpy(dX, this->x.data(), BYTES, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuSequential<T>::copyD2H()
{
    //gpuMemcpy(this->y.data(), dY, BYTES, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuSequential<T>::~GpuSequential()
{
    gpuFree(dX);
    gpuFree(dY);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuSequential<T>::solver()
{
    deviceAllocations();
 //   copyH2D();
 //   gpuSequential<T> << < GRID_SIZE, BLOCK_SIZE >> > (dX, dY, MEAN, SIGMA, N);
 //   gpuCheckErrors("gpu kernel launch failure");
 //   copyD2H();
}

template void GpuSequential<float>::solver();
template void GpuSequential<double>::solver();
template void GpuSequential<float>::deviceAllocations();
template void GpuSequential<double>::deviceAllocations();
template void GpuSequential<float>::copyH2D();
template void GpuSequential<double>::copyH2D();
template GpuSequential<float>::~GpuSequential();
template GpuSequential<double>::~GpuSequential();
