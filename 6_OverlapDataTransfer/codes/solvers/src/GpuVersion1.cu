#include "hip/hip_runtime.h"
#include "../include/GpuVersion1.h"

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuVersion1(T* a, int offset)
{
    int idx = offset + blockDim.x * blockIdx.x + threadIdx.x;
    T iX = static_cast<T>(idx);
    T s = sin(iX);
    T c = cos(iX);
    a[idx] = sqrt(s * s + c * c);
}

template<typename T>
void GpuVersion1<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dA, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuVersion1<T>::copyH2D(size_t offset, gpuStream_t stream)
{
    gpuMemcpyAsync(&dA[offset], &a[offset], STREAM_BYTES, gpuMemcpyHostToDevice, stream);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuVersion1<T>::copyD2H(size_t offset, gpuStream_t stream)
{
    gpuMemcpyAsync(&a[offset], &dA[offset], STREAM_BYTES, gpuMemcpyDeviceToHost, stream);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

/*
template<typename T>
void GpuVersion1<T>::launchSetup()
{
    auto blocksPerSM = 2048 / BLOCK_SIZE;
    int devID;
    int numSMs;
    gpuGetDevice(&devID);

    gpuDeviceGetAttribute(&numSMs, gpuDevAttrMultiProcessorCount, devID);
    std::cout << "There are " << numSMs << " SMs in this device." << std::endl;
    std::cout << "Blocks per SM: " << blocksPerSM << std::endl;

    gridSize = blocksPerSM * numSMs;
    std::cout << "Grid Size: " << gridSize << std::endl;
    std::cout << "Block Size: " << BLOCK_SIZE << std::endl;
}
*/

template<typename T>
GpuVersion1<T>::~GpuVersion1()
{
    gpuFree(dA);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuVersion1<T>::solver()
{
    int device;
    gpuGetDevice(&device);
    gpuDeviceProp_t devProp;
    gpuGetDeviceProperties(&devProp, device);
    deviceAllocations();
    
    // Stream setup
    //gpuEvent_t startEvent, stopEvent;
    gpuEvent_t dummyEvent;
    gpuStream_t stream[N_STREAMS];
    //gpuEventCreate(&startEvent);
    //gpuEventCreate(&stopEvent);
    gpuEventCreate(&dummyEvent);
    gpuCheckErrors("event create failure");

    for (int i = 0; i < N_STREAMS; ++i)
    {
        gpuStreamCreate(&stream[i]);
        gpuCheckErrors("stream create failure");
    }
    
    //launchSetup();

    // VERSION 1 algorithm
    //gpuEventRecord(startEvent, 0);
    //gpuCheckErrors("event record failure");
    for (int i = 0; i < N_STREAMS; ++i)
    {
        int offset = i * STREAM_SIZE;
        copyH2D(offset, stream[i]);
        gpuVersion1 << < GRID_SIZE/N_STREAMS, BLOCK_SIZE, 0, stream[i] >> > (dA, offset);
        copyD2H(offset, stream[i]);
    }
    gpuEventRecord(dummyEvent, 0);
    gpuCheckErrors("event record failure");
    gpuEventSynchronize(dummyEvent);
    gpuCheckErrors("event sync failure");
    //gpuEventElapsedTime(&ms, startEvent, stopEvent);
    //gpuCheckErrors("event elapsed time failure");
    //cout << "Version2 passed time in ms: " << ms << endl;

    // Cleanup
    //gpuEventDestroy(startEvent);
    //gpuEventDestroy(stopEvent);
    gpuEventDestroy(dummyEvent);
    for (int i = 0; i < N_STREAMS; ++i)
    {
        gpuStreamDestroy(stream[i]);
        gpuCheckErrors("stream destroy failure");
    }
    
}

template void GpuVersion1<float>::solver();
template void GpuVersion1<double>::solver();
template void GpuVersion1<float>::deviceAllocations();
template void GpuVersion1<double>::deviceAllocations();
template void GpuVersion1<float>::copyH2D(size_t, gpuStream_t);
template void GpuVersion1<double>::copyH2D(size_t, gpuStream_t);
template void GpuVersion1<float>::copyD2H(size_t, gpuStream_t);
template void GpuVersion1<double>::copyD2H(size_t, gpuStream_t);
template GpuVersion1<float>::~GpuVersion1();
template GpuVersion1<double>::~GpuVersion1();
