#include "hip/hip_runtime.h"
#include "../include/GpuSolver2.h"

#ifdef KERNELTIME
#include <omp.h>
#endif

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuSolver2(T* __restrict__ v, T* __restrict__ y)
{
    __shared__ T tile[TILE_DIM][TILE_DIM];

    int ix = blockIdx.x * TILE_DIM + threadIdx.x;
    int iy = blockIdx.y * TILE_DIM + threadIdx.y;
    //int width = gridDim.x * TILE_DIM;
    int width = N;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        tile[threadIdx.y + j][threadIdx.x] = v[(iy + j) * width + ix];
    }

    __syncthreads();

    ix = blockIdx.y * TILE_DIM + threadIdx.x;
    iy = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        y[(iy + j) * width + ix] = tile[threadIdx.x][threadIdx.y + j];
    }
    

}

template<typename T>
void GpuSolver2<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dV, BYTES);
    gpuMalloc(&dY, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuSolver2<T>::copyH2D()
{
    gpuMemcpy(dV, this->v.data(), BYTES, gpuMemcpyHostToDevice);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuSolver2<T>::copyD2H()
{
    gpuMemcpy(this->y.data(), dY, BYTES, gpuMemcpyDeviceToHost);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuSolver2<T>::~GpuSolver2()
{
    gpuFree(dV);
    gpuFree(dY);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuSolver2<T>::solver()
{

    deviceAllocations();

    copyH2D();
    dim3 threads(TILE_DIM, BLOCK_ROWS, 1);
    dim3 blocks(N / TILE_DIM, N / TILE_DIM, 1);
#ifdef KERNELTIME
    auto t0 = omp_get_wtime();
    gpuSolver2<T> << < blocks, threads >> > (dV, dY);
    gpuCheckErrors("gpu kernel launch failure");
    gpuDeviceSynchronize();
    auto t1 = omp_get_wtime();
    cout << "Kernel runtime: " << (t1 - t0) * 1000.0 << " ms." << endl;
#else
    gpuSolver2<T> << < blocks, threads >> > (dV, dY);
    gpuCheckErrors("gpu kernel launch failure");
#endif
    copyD2H();

}

template void GpuSolver2<float>::solver();
template void GpuSolver2<double>::solver();
template void GpuSolver2<float>::deviceAllocations();
template void GpuSolver2<double>::deviceAllocations();
template void GpuSolver2<float>::copyH2D();
template void GpuSolver2<double>::copyH2D();
template GpuSolver2<float>::~GpuSolver2();
template GpuSolver2<double>::~GpuSolver2();
