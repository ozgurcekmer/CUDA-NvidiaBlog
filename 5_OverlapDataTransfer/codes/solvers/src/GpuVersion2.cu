#include "hip/hip_runtime.h"
#include "../include/GpuVersion2.h"

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuVersion2(T* a, int offset)
{
    int idx = offset + blockDim.x * blockIdx.x + threadIdx.x;
    T iX = static_cast<T>(idx);
    T s = sin(iX);
    T c = cos(iX);
    a[idx] = sqrt(s * s + c * c);
}

template<typename T>
void GpuVersion2<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dA, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuVersion2<T>::copyH2D(size_t offset, gpuStream_t stream)
{
    gpuMemcpyAsync(&dA[offset], &a[offset], STREAM_BYTES, gpuMemcpyHostToDevice, stream);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuVersion2<T>::copyD2H(size_t offset, gpuStream_t stream)
{
    gpuMemcpyAsync(&a[offset], &dA[offset], STREAM_BYTES, gpuMemcpyDeviceToHost, stream);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuVersion2<T>::~GpuVersion2()
{
    gpuFree(dA);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuVersion2<T>::solver()
{
    deviceAllocations();

    // Stream setup
    gpuEvent_t startEvent, stopEvent;
    gpuStream_t stream[N_STREAMS];
   
    gpuEventCreate(&startEvent);
    gpuEventCreate(&stopEvent);
    gpuCheckErrors("event create failure");

    for (int i = 0; i < N_STREAMS; ++i)
    {
        gpuStreamCreate(&stream[i]);
        gpuCheckErrors("stream create failure");
    }

    // VERSION 2 algorithm
    gpuEventRecord(startEvent, 0);
    gpuCheckErrors("event record failure");
    for (int i = 0; i < N_STREAMS; ++i)
    {
        int offset = i * STREAM_SIZE;
        copyH2D(offset, stream[i]);
    }
    for (int i = 0; i < N_STREAMS; ++i)
    {
        int offset = i * STREAM_SIZE;
        gpuVersion2 << < GRID_SIZE / N_STREAMS, BLOCK_SIZE, 0, stream[i] >> > (dA, offset);
    }
    for (int i = 0; i < N_STREAMS; ++i)
    {
        int offset = i * STREAM_SIZE;
        copyD2H(offset, stream[i]);
    }
    gpuEventRecord(stopEvent, 0);
    gpuCheckErrors("event record failure");
    gpuEventSynchronize(stopEvent);
    gpuCheckErrors("event sync failure");
    gpuEventElapsedTime(&ms, startEvent, stopEvent);
    gpuCheckErrors("event elapsed time failure");
    cout << "Version2 passed time in ms: " << ms << endl;

    // Cleanup
    gpuEventDestroy(startEvent);
    gpuEventDestroy(stopEvent);
    for (int i = 0; i < N_STREAMS; ++i)
    {
        gpuStreamDestroy(stream[i]);
        gpuCheckErrors("stream destroy failure");
    }

}

template void GpuVersion2<float>::solver();
template void GpuVersion2<double>::solver();
template void GpuVersion2<float>::deviceAllocations();
template void GpuVersion2<double>::deviceAllocations();
template void GpuVersion2<float>::copyH2D(size_t, gpuStream_t);
template void GpuVersion2<double>::copyH2D(size_t, gpuStream_t);
template void GpuVersion2<float>::copyD2H(size_t, gpuStream_t);
template void GpuVersion2<double>::copyD2H(size_t, gpuStream_t);
template GpuVersion2<float>::~GpuVersion2();
template GpuVersion2<double>::~GpuVersion2();
