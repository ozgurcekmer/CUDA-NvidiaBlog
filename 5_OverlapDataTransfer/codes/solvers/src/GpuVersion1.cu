#include "hip/hip_runtime.h"
#include "../include/GpuVersion1.h"

using std::cout;
using std::endl;
using std::vector;

template <typename T>
__global__
void gpuVersion1(T* a, int offset)
{
    int idx = offset + blockDim.x * blockIdx.x + threadIdx.x;
    
    T iX = static_cast<T>(idx);
    T s = sin(iX);
    T c = cos(iX);
    a[idx] = sqrt(s * s + c * c);
}

template<typename T>
void GpuVersion1<T>::deviceAllocations()
{
    // Allocate device vectors
    gpuMalloc(&dA, BYTES);
    gpuCheckErrors("gpuMalloc failure");
}

template<typename T>
void GpuVersion1<T>::copyH2D(size_t offset, gpuStream_t stream)
{
    gpuMemcpyAsync(&dA[offset], &a[offset], STREAM_BYTES, gpuMemcpyHostToDevice, stream);
    gpuCheckErrors("gpuMemcpy H2D failure");
}

template<typename T>
void GpuVersion1<T>::copyD2H(size_t offset, gpuStream_t stream)
{
    gpuMemcpyAsync(&a[offset], &dA[offset], STREAM_BYTES, gpuMemcpyDeviceToHost, stream);
    gpuCheckErrors("gpuMemcpy D2H failure");
}

template<typename T>
GpuVersion1<T>::~GpuVersion1()
{
    gpuFree(dA);
    gpuCheckErrors("gpuFree failure");
}

template <typename T>
void GpuVersion1<T>::solver()
{
    deviceAllocations();
    
    // Stream setup
    gpuEvent_t startEvent, stopEvent;
    gpuStream_t stream[N_STREAMS];
    gpuEventCreate(&startEvent);
    gpuEventCreate(&stopEvent);
    gpuCheckErrors("event create failure");

    for (int i = 0; i < N_STREAMS; ++i)
    {
        gpuStreamCreate(&stream[i]);
        gpuCheckErrors("stream create failure");
    }
    
    // VERSION 1 algorithm
    gpuEventRecord(startEvent, 0);
    gpuCheckErrors("event record failure");
    for (int i = 0; i < N_STREAMS; ++i)
    {
        int offset = i * STREAM_SIZE;
        copyH2D(offset, stream[i]);
        gpuVersion1 << < GRID_SIZE/N_STREAMS, BLOCK_SIZE, 0, stream[i] >> > (dA, offset);
        copyD2H(offset, stream[i]);
        gpuStreamQuery(stream[i]);
    }
  
    gpuEventRecord(stopEvent, 0);
    gpuCheckErrors("event record failure");
    gpuEventSynchronize(stopEvent);
    gpuCheckErrors("event sync failure");
    gpuEventElapsedTime(&ms, startEvent, stopEvent);
    gpuCheckErrors("event elapsed time failure");
    cout << "Version1 passed time in ms: " << ms << endl;

    // Cleanup
    gpuEventDestroy(startEvent);
    gpuEventDestroy(stopEvent);
    for (int i = 0; i < N_STREAMS; ++i)
    {
        gpuStreamDestroy(stream[i]);
        gpuCheckErrors("stream destroy failure");
    }
    
}

template void GpuVersion1<float>::solver();
template void GpuVersion1<double>::solver();
template void GpuVersion1<float>::deviceAllocations();
template void GpuVersion1<double>::deviceAllocations();
template void GpuVersion1<float>::copyH2D(size_t, gpuStream_t);
template void GpuVersion1<double>::copyH2D(size_t, gpuStream_t);
template void GpuVersion1<float>::copyD2H(size_t, gpuStream_t);
template void GpuVersion1<double>::copyD2H(size_t, gpuStream_t);
template GpuVersion1<float>::~GpuVersion1();
template GpuVersion1<double>::~GpuVersion1();
